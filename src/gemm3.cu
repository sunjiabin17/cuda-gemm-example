#include <hip/hip_runtime.h>

#include <algorithm>
#include <random>
#include <vector>

template <typename T, size_t BLOCK_TILE_SIZE_N, size_t BLOCK_TILE_SIZE_M,
          size_t BLOCK_TILE_SIZE_K, size_t NUM_THREADS,
          size_t BLOCK_TILE_SKEW_SIZE_X = 0U,
          size_t BLOCK_TILE_SKEW_SIZE_K = 0U>
__device__ void load_data_to_shared_memory(
    const T *A, size_t lda, const T *B, size_t ldb,
    T A_shared_memory_block[BLOCK_TILE_SIZE_M]
                           [BLOCK_TILE_SIZE_K + BLOCK_TILE_SKEW_SIZE_K],
    T B_shared_memory_block[BLOCK_TILE_SIZE_K]
                           [BLOCK_TILE_SIZE_N + BLOCK_TILE_SKEW_SIZE_K],
    size_t shared_memory_block_tile_idx, size_t thread_linear_idx, size_t m,
    size_t n, size_t k) {

  // load A
  const size_t a_load_num =
      (BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K + NUM_THREADS - 1U) / NUM_THREADS;
  for (size_t load_idx = 0U; load_idx < a_load_num; ++load_idx) {
    const size_t A_shared_memory_block_row_idx =
        (thread_linear_idx + load_idx * NUM_THREADS) / BLOCK_TILE_SIZE_K;
    const size_t A_shared_memory_block_col_idx =
        (thread_linear_idx + load_idx * NUM_THREADS) % BLOCK_TILE_SIZE_K;

    const size_t A_row_idx =
        blockIdx.y * BLOCK_TILE_SIZE_M + A_shared_memory_block_row_idx;
    // [TODO] 这里为什么是shared_memory_block_tile_idx，而不是blockIdx.x;
    const size_t A_col_idx = shared_memory_block_tile_idx * BLOCK_TILE_SIZE_K +
                             A_shared_memory_block_col_idx;

    T val = static_cast<T>(0);
    if (A_row_idx < m and A_col_idx < k) {
      val = A[A_row_idx * lda + A_col_idx];
    }
    static_assert(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K % NUM_THREADS == 0U);

    A_shared_memory_block[A_shared_memory_block_row_idx]
                         [A_shared_memory_block_col_idx] = val;
  }

  // load B
  size_t b_load_num =
      (BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_N + NUM_THREADS - 1) / NUM_THREADS;
  for (size_t load_idx = 0U; load_idx < b_load_num; ++load_idx) {
    const size_t B_shared_memory_block_row_idx =
        (thread_linear_idx + load_idx * NUM_THREADS) / BLOCK_TILE_SIZE_N;
    const size_t B_shared_memory_block_col_idx =
        (thread_linear_idx + load_idx * NUM_THREADS) % BLOCK_TILE_SIZE_N;

    const size_t B_row_idx = shared_memory_block_tile_idx * BLOCK_TILE_SIZE_K +
                             B_shared_memory_block_row_idx;
    const size_t B_col_idx =
        blockIdx.x * BLOCK_TILE_SIZE_N + B_shared_memory_block_col_idx;

    T val = static_cast<T>(0);
    if (B_row_idx < k and B_col_idx < n) {
      val = B[B_row_idx * ldb + B_col_idx];
    }
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_N % NUM_THREADS == 0U);

    B_shared_memory_block[B_shared_memory_block_row_idx]
                         [B_shared_memory_block_col_idx] = val;
  }
}

template <typename T, size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N,
          size_t BLOCK_TILE_SIZE_K>
__global__ void gemm_v3(size_t m, size_t n, size_t k, T alpha, const T *A,
                        size_t lda, const T *B, size_t ldb, T beta, T *C,
                        size_t ldc) {
  constexpr size_t NUM_THREADS = BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_N;
  const size_t thread_linear_idx = blockIdx.y * blockDim.x + threadIdx.x;

  const size_t C_row_idx = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t C_col_idx = blockDim.x * blockIdx.x + threadIdx.x;

  __shared__ T A_shared_memory_block[BLOCK_TILE_SIZE_M][BLOCK_TILE_SIZE_K];
  __shared__ T B_shared_memory_block[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_N];

  const size_t num_shared_memory_block_tile =
      (k + BLOCK_TILE_SIZE_K - 1) / BLOCK_TILE_SIZE_K;

  size_t sum = static_cast<T>(0);
  for (size_t shared_memory_block_tile_idx = 0U;
       shared_memory_block_tile_idx < num_shared_memory_block_tile;
       ++shared_memory_block_tile_idx) {
    load_data_to_shared_memory<T, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_M,
                               BLOCK_TILE_SIZE_K, NUM_THREADS>(
        A, lda, B, ldb, A_shared_memory_block, B_shared_memory_block,
        shared_memory_block_tile_idx, thread_linear_idx, m, n, k);
    __syncthreads();

    for (size_t k_idx = 0U; k_idx < BLOCK_TILE_SIZE_K; ++k_idx) {
      sum += A_shared_memory_block[threadIdx.y][k_idx] *
             B_shared_memory_block[k_idx][threadIdx.x];
    }
    __syncthreads();
  }
  if (C_row_idx < m and C_col_idx < n) {
    C[C_row_idx * ldc + C_col_idx] =
        alpha * sum + beta * C[C_row_idx * ldc + C_col_idx];
  }
}

template <typename T>
void launch_gemm_v3(size_t m, size_t n, size_t k, T alpha, const T *A,
                    size_t lda, const T *B, size_t ldb, T beta, T *C,
                    size_t ldc, hipStream_t stream) {
  constexpr unsigned int BLOCK_TILE_SIZE_M = 32U;
  constexpr unsigned int BLOCK_TILE_SIZE_N = 32U;
  constexpr unsigned int BLOCK_TILE_SIZE_K = 32U;
  constexpr unsigned int NUM_THREADS = (BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_N);
  static_assert(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K % NUM_THREADS == 0U);
  static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_N % NUM_THREADS == 0U);
  const dim3 block_dim{BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_M, 1};
  const dim3 grid_dim{
      (static_cast<unsigned int>(n) + block_dim.x - 1) / block_dim.x,
      (static_cast<unsigned int>(m) + block_dim.y - 1) / block_dim.y, 1};
  gemm_v3<T, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K>
      <<<grid_dim, block_dim, 0, stream>>>(m, n, k, alpha, A, lda, B, ldb, beta,
                                           C, ldc);
}

using T = float;

int main(int argc, char **argv) {
  size_t m = 1024;
  size_t n = 1024;
  size_t k = 1024;

  T *A = new T[m * k];
  T *B = new T[k * n];
  T *C = new T[m * n];
  T *C1 = new T[m * n];

  // set random seed
  // srand((unsigned)time(NULL));

  std::generate(A, A + m * k, []() { return (T)(rand() % 10); });
  std::generate(B, B + k * n, []() { return (T)(rand() % 10); });
  std::fill(C, C + m * n, 0.0f);

  T *dA, *dB, *dC;
  hipMalloc(&dA, m * k * sizeof(T));
  hipMalloc(&dB, k * n * sizeof(T));
  hipMalloc(&dC, m * n * sizeof(T));

  hipMemcpy(dA, A, m * k * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(dB, B, k * n * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(dC, C, m * n * sizeof(T), hipMemcpyHostToDevice);

  T alpha = 1.f;
  T beta = 0.f;

  size_t lda = k;
  size_t ldb = n;
  size_t ldc = n;

  hipStream_t stream;
  hipStreamCreate(&stream);

  launch_gemm_v3(m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, stream);

  // [TODO] something

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  delete[] A;
  delete[] B;
  delete[] C;
  delete[] C1;

  return 0;
}